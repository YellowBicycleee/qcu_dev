#include "qcu_wilson_dslash.cuh"
#include "kernel/qcu_wilson_dslash_naive.cuh"
#include <chrono>
#include <cstdio>

namespace qcu {
  extern int proc_size[Nd];

  void WilsonDslash::calculateDslash(int dagger_flag) {

  }


  void WilsonDslash::calculateDslashNaive(int dagger_flag) {

    int Lx = dslashParam_->Lx;
    int Ly = dslashParam_->Ly;
    int Lz = dslashParam_->Lz;
    int Lt = dslashParam_->Lt;
    int parity = dslashParam_->parity;
    double flag = (dagger_flag == 0) ? 1.0 : -1.0;

    int half_vol = Lx * Ly * Lz * Lt >> 1;
    int block_size = BLOCK_SIZE;
    int grid_size = (half_vol + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 gridDim(grid_size);
    dim3 blockDim(block_size);
  
    qcuCudaDeviceSynchronize();
  
    // mpi_comm->preDslash(dslashParam_->fermion_in, parity, invert_flag);
  
    auto start = std::chrono::high_resolution_clock::now();

    void *args[] = {&dslashParam_->gauge, &dslashParam_->fermion_in, \
                    &dslashParam_->fermion_out, &Lx, &Ly, &Lz, &Lt, \
                    &parity, &proc_size[0], &proc_size[1], &proc_size[2], \
                    &proc_size[3], &flag};
  
    checkCudaErrors(hipLaunchKernel((void *)mpiDslashNaive, gridDim, blockDim, args));

    qcuCudaDeviceSynchronize();

    // boundary calculate
    // mpi_comm->postDslash(dslashParam_->fermion_out, parity, invert_flag);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    printf("naive total time: (without malloc free memcpy) : %.9lf sec, block size = %d\n", double(duration) / 1e9, block_size);
  }


  void callWilsonDslashNaive(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity, int dagger_flag) {
    printf("Naive Wilson Dslash...\n");
    DslashParam dslash_param(fermion_in, fermion_out, gauge, param, parity);
    WilsonDslash dslash_solver(dslash_param);
    dslash_solver.calculateDslashNaive(dagger_flag);
  }

  void callFullWilsonDslashNaive(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity, int dagger_flag, double kappa) {

    int Lx = param->lattice_size[0];
    int Ly = param->lattice_size[1];
    int Lz = param->lattice_size[2];
    int Lt = param->lattice_size[3];
    int half_vol = Lx / 2 * Ly * Lz * Lt;
    void* diag_fermion_in = static_cast<void*>(static_cast<Complex*>(fermion_in) + parity * half_vol * Ns * Nc);
    void* non_diag_fermion_in = static_cast<void*>(static_cast<Complex*>(fermion_in) + (1-parity) * half_vol * Ns * Nc);

    DslashParam dslash_param(non_diag_fermion_in, fermion_out, gauge, param, parity);
    WilsonDslash dslash_solver(dslash_param);
    dslash_solver.calculateDslashNaive(dagger_flag);


    // dst = src - kappa dst

    int block_size = BLOCK_SIZE;
    int grid_size = (half_vol + block_size - 1) / block_size;
    mpiDslashNaiveTail<<<grid_size, block_size>>>(gauge, diag_fermion_in, fermion_out, Lx, Ly, Lz, Lt, parity, kappa);
    qcuCudaDeviceSynchronize();
  }
};
